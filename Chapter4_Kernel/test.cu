// CUDA hello, world
#include <iostream>
#include <hip/hip_runtime.h>


__global__ void helloFromGPU() {
    printf("Hello World from GPU!\n");
}

int main() {
    // Hello from CPU
    printf("Hello World from CPU!\n");

    // Hello from GPU
    helloFromGPU<<<2, 2>>>();
    hipDeviceReset();

    return 0;
}

